#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <typeinfo>
#include "ElectricField.cuh"

/*
__global__ void d_getEfield(ElectricField *meshfield, 
														std::vector<std::vector<double> > *samplePoints,
														std::vector<double> *eValues)
{
	int id = threadIdx.x;
	std::vector<double> thisPoint(2, 0);
	thisPoint[0] = 3.; //samplePoints[id][0];
	thisPoint[1] = 10.;  //samplePoints[id][1];
	meshfield->getEvalue(&thisPoint, &eValues);
	eValues[id] = meshfield->getEvalue(thisPoint);
}
*/

int main()
{
	// Create E_field
	srand ( time(NULL) );
	ElectricField eField(0., 10., 2., 24., 100, 100);
	ElectricField* h_p_eField = &eField;
	ElectricField *d_p_eField;	
	// Initialize sample points
	std::vector< std::vector<double> > h_samples(20, std::vector<double>(2, 0));
	for(int i=0; i<20; i++){
		h_samples.at(i).at(0) = rand() % 10;
		h_samples.at(i).at(1) = rand() % 22 + 2;
	}
	std::cout << eField.get_grid().at(0).size() << std::endl;
	std::vector< std::vector<double> > *d_samplesPtr;
	std::vector< double > h_result(20, 0), *d_values;
	
	int sampleVct_size = h_samples.size() * h_samples[0].size() * sizeof(double); //usefull number
	int resultVct_size = h_result.size() * sizeof(double); //usefull number

	// Get device memory ready
	hipMalloc((void **) &d_samplesPtr, sampleVct_size);
	hipMemcpy((void *) d_samplesPtr, (void *) &h_samples, sampleVct_size, hipMemcpyHostToDevice);
	hipMalloc((void **) &d_values, resultVct_size);
	hipMemcpy((void *) d_values, (void *) &h_result, resultVct_size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_p_eField, sizeof(ElectricField));	
	hipMemcpy((void *)d_p_eField, (void *) h_p_eField, sizeof(ElectricField), hipMemcpyHostToDevice);

	for(int j=0; j<20; j++){
	std::cout << h_samples.at(j).at(0) << std::endl;
	std::cout << eField.getEvalue(h_samples.at(j)) << std::endl;
	} // On host

	// clean device
	hipFree((void *) d_samplesPtr);
	hipFree((void *) d_p_eField);
	hipFree((void *) d_values);
	return 0;
}
